
#include <hip/hip_runtime.h>
void vecAddTraditional(float *A_h, float *B_h, float *C_h, int n)
{
    for (int i = 0; i < n; ++i)
    {
        C_h[i] = A_h[i] + B_h[i];
    }
}

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

void vecAddCuda(float *A_h, float *B_h, float *C_h, int n)
{
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // kernel code invocation code
    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
    return 0;
}