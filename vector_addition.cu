
#include <hip/hip_runtime.h>
void vecAddTraditional(float* A_h, float* B_h, float* C_h, int n) {
    for (int i =0; i < n; ++i) {
        C_h[i] = A_h[i] + B_h[i];
    }
}


void vecAddCuda(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);
    
    // kernel code invocation code
    // ...
    
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);


    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}


int main() {
    return 0;
}